/*
This program is written to find the nearest neighbour of each point in 3 deminsional
space by implementing the brute force algorithm.
The brute force approach can easily be converted into a embarassingly parallel algorithm for
the GPU where there is no interaction between the threads.
Benchmarking is done to compare the CPU and GPU computational approaches to the problem.
*/

/*
Note that there is a considerable dependency of the ratio of execution times of the CPU and GPU on the 
hardware which is being used to execute the run the program.
*/

// Importing the required headers
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<time.h>

struct position
{
    int x,y,z;      //odd number of parameters in the structure helps reducing bank conflicts in shared memory(if used).
};


// Returns the duration from start to end times in sec
double time_elapsed(struct timespec *start, struct timespec *end) 
{
	double t;
	t = (end->tv_sec - start->tv_sec); // diff in seconds
	t += (end->tv_nsec - start->tv_nsec) * 0.000000001; //diff in nanoseconds
	return t;
}

// GPU Kernel
__global__ void GPU_Find(struct position *points, int *nearest, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int min = 1<<22;
    int temp;

    if(i >= n) return;

    for(int j = 0; j < n; j++)
    {
        if(i == j) continue;

        temp = (points[i].x - points[j].x)*(points[i].x - points[j].x); 
        temp += (points[i].y - points[j].y)*(points[i].y - points[j].y);
        temp += (points[i].z - points[j].z)*(points[i].z - points[j].z);

        if(temp < min)
        {
            min = temp;
            nearest[i] = j;
        }
    }

    return;
}

// CPU Function
void CPU_Find(struct position *points, int *nearest, int n)
{
    int min;       //All the distances are going to be smaller than this.
    int temp;

    for(int i = 0; i < n; i++)
    {
        min = 1<<22;
        for(int j = 0; j < n; j++)
        {
            if(i == j) continue;

            temp = (points[i].x - points[j].x)*(points[i].x - points[j].x); 
            temp += (points[i].y - points[j].y)*(points[i].y - points[j].y);
            temp += (points[i].z - points[j].z)*(points[i].z - points[j].z);

            temp = (int)sqrt(temp);

            if(temp < min)
            {
                min = temp;
                nearest[i] = j;
            }
        }
    }

    return;
}

// Code execution begins here
int main()
{
    struct timespec start1, end1;
    struct timespec start2, end2;

    int n;

    printf("Enter the value of n: ");
    scanf("%d", &n);

    struct position *points;
    int *nearest1;
    int *nearest2;

    hipMallocManaged(&points, n*sizeof(struct position));
    hipMallocManaged(&nearest1, n*sizeof(int));
    hipMallocManaged(&nearest2, n*sizeof(int));

    for(int i = 0; i < n; i++)
    {
        points[i].x = rand()%100000;
        points[i].y = rand()%100000;
        points[i].z = rand()%10000;
        nearest1[i] = -1;
        nearest2[i] = -1;
    }

    clock_gettime(CLOCK_REALTIME, &start1); //start timestamp
	GPU_Find<<<(n/128+1), 128>>>(points, nearest1, n);
	hipDeviceSynchronize();
    clock_gettime(CLOCK_REALTIME, &end1);	//end timestamp
    
    clock_gettime(CLOCK_REALTIME, &start2); //start timestamp
	CPU_Find(points, nearest2, n);
    clock_gettime(CLOCK_REALTIME, &end2);	//end timestamp
    
    printf("\nTime taken by GPU is: %lf\n", time_elapsed(&start1, &end1));	 //print result for GPU
    printf("Time taken by CPU is: %lf\n", time_elapsed(&start2, &end2));	 //print result for CPU

    hipFree(points);
    hipFree(nearest1);
    hipFree(nearest2);

    return 0;
}

/*
The results obtained by the CPU and GPU may differ. Why so?
*/
